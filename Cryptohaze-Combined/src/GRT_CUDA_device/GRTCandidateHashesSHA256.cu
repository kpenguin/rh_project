#include "hip/hip_runtime.h"
// This is here so Netbeans doesn't error-spam my IDE
#if !defined(__HIPCC__)
    // define the keywords, so that the IDE does not complain about them
    #define __global__
    #define __device__
    #define __shared__
    #define __constant__
    #define blockIdx.x 1
    #define blockDim.x 1
    #define threadIdx.x 1
#endif

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include "../../inc/GRT_Common/GRTCommon.h"



// These will be the same for all GPUs working on a hash.
__device__ __constant__ unsigned char SHA256_Candidate_Device_Charset_Constant[512]; // Constant space for charset
__device__ __constant__ uint32_t SHA256_Candidate_Device_Charset_Length; // Character set length
__device__ __constant__ uint32_t SHA256_Candidate_Device_Chain_Length; // May as well pull it from constant memory... faster when cached.
__device__ __constant__ uint32_t SHA256_Candidate_Device_Table_Index;
__device__ __constant__ uint32_t SHA256_Candidate_Device_Number_Of_Threads; // It needs this, and can't easily calculate it

// 4 32-byte words for SHA256 hashes
__device__ __constant__ uint32_t SHA256_Candidate_Device_Hash[5];


#include "../../inc/CUDA_Common/CUDA_SHA256.h"
#include "../../inc/CUDA_Common/Hash_Common.h"
#include "../../inc/GRT_CUDA_device/CUDA_Reduction_Functions.h"

// Copy the shared variables to the host
extern "C" void copySHA256CandidateDataToConstant(char *hostCharset, uint32_t hostCharsetLength,
        uint32_t hostChainLength, uint32_t hostTableIndex, uint32_t hostNumberOfThreads) {

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("SHA256_Candidate_Device_Charset_Constant"), hostCharset, 512));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("SHA256_Candidate_Device_Charset_Length"), &hostCharsetLength, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("SHA256_Candidate_Device_Chain_Length"), &hostChainLength, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("SHA256_Candidate_Device_Table_Index"), &hostTableIndex, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("SHA256_Candidate_Device_Number_Of_Threads"), &hostNumberOfThreads, sizeof(uint32_t)));
}


extern "C" void copySHA256HashDataToConstant(unsigned char *hash) {
    // Yes, I'm copying into a uint32_t array from an unsigned char array.  This works, though, and it makes
    // my life easier.
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("SHA256_Candidate_Device_Hash"), hash, 20 * sizeof(unsigned char)));
}


#define CREATE_SHA256_CH_KERNEL(length) \
__global__ void GenerateSHA256CH##length(unsigned char *CandidateHashes, uint32_t ThreadSpaceOffset, uint32_t StartStep, uint32_t StepsToRun) { \
    const int pass_length = length; \
    uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
    uint32_t a,b,c,d,e, f, g, h; \
    uint32_t *InitialArray32; \
    uint32_t *OutputArray32; \
    InitialArray32 = (uint32_t *)SHA256_Candidate_Device_Hash; \
    OutputArray32 = (uint32_t *)CandidateHashes; \
    uint32_t i, chain_index, step_to_calculate, charset_offset, last_step_for_iteration; \
    __shared__ char charset[512]; \
    copySingleCharsetToShared(charset, SHA256_Candidate_Device_Charset_Constant); \
    chain_index = ((blockIdx.x*blockDim.x + threadIdx.x) + (ThreadSpaceOffset * SHA256_Candidate_Device_Number_Of_Threads)); \
    if ((chain_index + StartStep) > SHA256_Candidate_Device_Chain_Length) { \
        return; \
    } \
    if (StartStep == 0) { \
        a = InitialArray32[0]; \
        b = InitialArray32[1]; \
        c = InitialArray32[2]; \
        d = InitialArray32[3]; \
        e = InitialArray32[4]; \
        f = InitialArray32[5]; \
        g = InitialArray32[6]; \
        h = InitialArray32[7]; \
    } else { \
        a = OutputArray32[0 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        b = OutputArray32[1 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        c = OutputArray32[2 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        d = OutputArray32[3 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        e = OutputArray32[4 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        f = OutputArray32[5 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        g = OutputArray32[6 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
        h = OutputArray32[7 * SHA256_Candidate_Device_Chain_Length + chain_index]; \
    } \
    step_to_calculate = chain_index + StartStep; \
    charset_offset = step_to_calculate % SHA256_Candidate_Device_Charset_Length; \
    clearB0toB15(b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
    reduceSingleCharsetNormal(b0, b1, b2, a, b, c, d, step_to_calculate, charset, charset_offset, pass_length, SHA256_Candidate_Device_Table_Index); \
    step_to_calculate++; \
    charset_offset++; \
    if (charset_offset >= SHA256_Candidate_Device_Charset_Length) { \
        charset_offset = 0; \
    } \
    if ((step_to_calculate + StepsToRun) > SHA256_Candidate_Device_Chain_Length) { \
        last_step_for_iteration = SHA256_Candidate_Device_Chain_Length - 1; \
    } else { \
        last_step_for_iteration = (step_to_calculate + StepsToRun - 1); \
    } \
    for (i = step_to_calculate; i <= last_step_for_iteration; i++) { \
        b15 = pass_length * 8; \
        SetCharacterAtPosition(0x80, pass_length, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15 ); \
		b0 = reverse(b0); b1 = reverse(b1); b2 = reverse(b2); b3 = reverse(b3); b4 = reverse(b4); \
        SHA256_FIRST_BLOCK(); \
        a = reverse(a);b = reverse(b);c = reverse(c);d = reverse(d);e = reverse(e); \
        clearB0toB15(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15); \
        reduceSingleCharsetNormal(b0, b1, b2, a, b, c, d, i, charset, charset_offset, pass_length, SHA256_Candidate_Device_Table_Index); \
        charset_offset++; \
        if (charset_offset >= SHA256_Candidate_Device_Charset_Length) { \
            charset_offset = 0; \
        } \
    } \
    OutputArray32[0 * SHA256_Candidate_Device_Chain_Length + chain_index] = a; \
    OutputArray32[1 * SHA256_Candidate_Device_Chain_Length + chain_index] = b; \
    OutputArray32[2 * SHA256_Candidate_Device_Chain_Length + chain_index] = c; \
    OutputArray32[3 * SHA256_Candidate_Device_Chain_Length + chain_index] = d; \
    OutputArray32[4 * SHA256_Candidate_Device_Chain_Length + chain_index] = e; \
    OutputArray32[5 * SHA256_Candidate_Device_Chain_Length + chain_index] = f; \
    OutputArray32[6 * SHA256_Candidate_Device_Chain_Length + chain_index] = g; \
    OutputArray32[7 * SHA256_Candidate_Device_Chain_Length + chain_index] = h; \
}

CREATE_SHA256_CH_KERNEL(6)
CREATE_SHA256_CH_KERNEL(7)
CREATE_SHA256_CH_KERNEL(8)
CREATE_SHA256_CH_KERNEL(9)
CREATE_SHA256_CH_KERNEL(10)


extern "C" void LaunchSHA256CandidateHashKernel(int PasswordLength, int CUDA_Blocks, int CUDA_Threads,
        unsigned char *DEVICE_End_Hashes, uint32_t ThreadSpaceOffset, uint32_t StartStep, uint32_t StepsToRun) {

    switch (PasswordLength) {
        case 6:
            GenerateSHA256CH6 <<< CUDA_Blocks, CUDA_Threads >>>
                (DEVICE_End_Hashes, ThreadSpaceOffset, StartStep, StepsToRun);
            break;
        case 7:
            GenerateSHA256CH7 <<< CUDA_Blocks, CUDA_Threads >>>
                (DEVICE_End_Hashes, ThreadSpaceOffset, StartStep, StepsToRun);
            break;
        case 8:
            GenerateSHA256CH8 <<< CUDA_Blocks, CUDA_Threads >>>
                (DEVICE_End_Hashes, ThreadSpaceOffset, StartStep, StepsToRun);
            break;
        case 9:
            GenerateSHA256CH9 <<< CUDA_Blocks, CUDA_Threads >>>
                (DEVICE_End_Hashes, ThreadSpaceOffset, StartStep, StepsToRun);
            break;
        case 10:
            GenerateSHA256CH10 <<< CUDA_Blocks, CUDA_Threads >>>
                (DEVICE_End_Hashes, ThreadSpaceOffset, StartStep, StepsToRun);
            break;
        default:
            printf("Password length %d not supported!", PasswordLength);
            exit(1);
    }
}
